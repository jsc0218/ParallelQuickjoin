#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include "QuickJoin.h"
using namespace std;

const int blockSize = 32;
const int gridSizeX = 10000;

#ifdef VECTOR
const string DBPath = "I:\\colors_112_112682.ascii";
const int DBSize = 112682;
__host__ Object *Read(istream& in)
{
	string cmdLine;
	double *x = new double[dimension];
	for (int i=0; i<dimension; ++i) 
	{
		in>>cmdLine;
		x[i] = atof(cmdLine.c_str());
	}
	Object *obj = new Object(x);
	delete[] x;
	return obj;
}
#endif

#ifdef STRING
const string DBPath = "I:\\English.dic";
const int DBSize = 69069;
__host__ Object *Read(istream& in)
{
	string cmdLine;
	getline(in, cmdLine);
	Object *obj = new Object(cmdLine.c_str(), (int)cmdLine.size());
	return obj;
}
#endif

__host__ int main(int argc, char** argv) 
{
	Object *h_objs = new Object[DBSize];
	ifstream fin(DBPath.c_str());
	for (int i=0; i<DBSize; ++i) 
	{
		Object *obj = Read(fin);
		h_objs[i] = *obj;
		delete obj;
	}
	fin.close();

	findCudaDevice(argc, (const char **)argv);
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	
	Object *d_objs;
	checkCudaErrors(hipMalloc(&d_objs, sizeof(Object)*DBSize));
	for (int i=0; i<DBSize; ++i)
	{
		#ifdef VECTOR
		double *tmp;
		checkCudaErrors(hipMalloc(&tmp, sizeof(double)*dimension));
		checkCudaErrors(hipMemcpy(tmp, h_objs[i].x, sizeof(double)*dimension, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(&(d_objs[i].x), &tmp, sizeof(double *), hipMemcpyHostToDevice));
		#endif
		
		#ifdef STRING
		char *tmp;
		checkCudaErrors(hipMalloc(&tmp, sizeof(char)*h_objs[i].length));
		checkCudaErrors(hipMemcpy(tmp, h_objs[i].x, sizeof(char)*h_objs[i].length, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(&(d_objs[i].x), &tmp, sizeof(char *), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(&(d_objs[i].length), &(h_objs[i].length), sizeof(int), hipMemcpyHostToDevice));
		#endif
	}
	delete[] h_objs;
	
	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	long long size = DBSize;
	long long mulSize = size * size;
	int blockNum = (int)ceil(mulSize / (float)blockSize);
	int gridSizeY = (int)ceil(blockNum / (float)gridSizeX);
	dim3 gridSize(gridSizeX, gridSizeY);
	NestedLoop<<<gridSize, blockSize, 0>>>(d_objs, 0, DBSize-1);
	checkCudaErrors(hipDeviceSynchronize());

	sdkStopTimer(&timer);
	cout<<"Processing time: "<<sdkGetTimerValue(&timer)<<" (ms)"<<endl;
	sdkDeleteTimer(&timer);
	
	checkCudaErrors(hipFree(d_objs));
	hipDeviceReset();
	exit(0);
}

